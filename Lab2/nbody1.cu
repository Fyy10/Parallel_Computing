#include "hip/hip_runtime.h"
#include "math.h"
#include "stdio.h"
#include "stdlib.h"
#include "timer.h"
#include "check.h"
#include "hip/hip_runtime.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct {
    float x, y, z, vx, vy, vz;
} Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float) RAND_MAX) - 1.0f;
    }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__ void bodyForceKernel(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f;
        float Fy = 0.0f;
        float Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            // r^2 = x^2 + y^2 + z^2
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            // 1 / r
            float invDist = rsqrtf(distSqr);
            // 1 / r^3
            float invDist3 = invDist * invDist * invDist;

            // F = GMm * r / r^3
            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        p[i].vx += dt * Fx;
        p[i].vy += dt * Fy;
        p[i].vz += dt * Fz;
    }
}

/*
 * This position integration cannot occur until this round of `bodyForce` has completed.
 * Also, the next round of `bodyForce` cannot begin until the integration is complete.
 */

__global__ void updatePositionKernel(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char **argv) {
    /*
     * Do not change the value for `nBodies` here. If you would like to modify it,
     * pass values into the command line.
     */

    int nBodies = 2 << 11;
    int salt = 0;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

#ifdef DEV
    #define GPUID 1
    // in the dev env
    // set gpu id
    hipSetDevice(GPUID);
#endif

    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */

    if (argc > 2) salt = atoi(argv[2]);

    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;

#ifdef UNIFIED
    // use unified memory
    hipMallocManaged((void **)&buf, bytes);
#else
    buf = (float *) malloc(bytes);
    // device mem
    Body *dev_p;
    hipMalloc((void **)&dev_p, bytes);
#endif
    Body *p = (Body *) buf;

    // num of threads
    size_t block_dim = 32;
    // num of blocks
    size_t grid_dim = (nBodies - 1) / block_dim + 1;

    /*
     * As a constraint of this exercise, `randomizeBodies` must remain a host function.
     */

    // 6 * nBodies of float in total
    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

#ifndef UNIFIED
    // copy from host to device
    hipMemcpy(dev_p, p, bytes, hipMemcpyHostToDevice);
#endif

    double totalTime = 0.0;

    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */

    /*******************************************************************/
    // Do not modify these 2 lines of code.
    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();
    /*******************************************************************/

        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */

#ifdef UNIFIED
        // compute interbody forces
        bodyForceKernel<<<grid_dim, block_dim>>>(p, dt, nBodies);

        // update positions
        updatePositionKernel<<<grid_dim, block_dim>>>(p, dt, nBodies);

        // if unified memory is used, device level synchronization at the last iter is required
        if (iter == nIters - 1) hipDeviceSynchronize();
#else
        // compute interbody forces
        bodyForceKernel<<<grid_dim, block_dim>>>(dev_p, dt, nBodies);

        // update positions
        updatePositionKernel<<<grid_dim, block_dim>>>(dev_p, dt, nBodies);

        // copy from device to host at the last iteration
        if (iter == nIters - 1) hipMemcpy(p, dev_p, bytes, hipMemcpyDeviceToHost);
#endif

    /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double) (nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/

    /*
     * Feel free to modify code below.
     */

#ifdef UNIFIED
    hipFree(buf);
#else
    free(buf);
    hipFree(dev_p);
#endif
}
