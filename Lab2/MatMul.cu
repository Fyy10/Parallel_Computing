#include "hip/hip_runtime.h"
#include "stdio.h"
#define TILE_WIDTH 10

void MatMul(float *A, float *B, float *C, int m, int n, int p);
__global__ void MatMulKernel(float *A, float *B, float *C, int m, int n, int p);
void printMat(float *A, int m, int n);

int main() {
    const int m = 10, n = 20, p = 10;
    float a[m][n], b[n][p], c[m][p];
    // generate A
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            a[i][j] = float(i + j);
        }
    }
    // generate B
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            b[i][j] = float(i + j);
        }
    }
    printf("Mat A:\n");
    printMat(*a, m, n);
    printf("Mat B:\n");
    printMat(*b, n, p);

    MatMul(*a, *b, *c, m, n, p);

    printf("Mat C:\n");
    printMat(*c, m, p);
    return 0;
}

void printMat(float *A, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.2f ", A[i * n + j]);
        }
        printf("\n");
    }
    printf("shape: (%d, %d)\n", m, n);
}

void MatMul(float *A, float *B, float *C, int m, int n, int p) {
    int sizeA = m * n * sizeof(float);
    int sizeB = n * p * sizeof(float);
    int sizeC = m * p * sizeof(float);
    float *dA, *dB, *dC;

    hipMalloc((void **)&dA, sizeA);
    hipMalloc((void **)&dB, sizeB);
    hipMalloc((void **)&dC, sizeC);
    hipMemcpy(dA, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeC, hipMemcpyHostToDevice);

    // 1 block
    dim3 DimGrid(1);
    // m * n threads
    dim3 DimBlock(m, p);

    MatMulKernel<<<DimGrid, DimBlock>>>(dA, dB, dC, m, n, p);

    hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}

__global__ void MatMulKernel(float *A, float *B, float *C, int m, int n, int p) {
    __shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * p + col;
    float value = 0.0;

    for (int phase = 0; phase < n / TILE_WIDTH; phase++) {
        s_A[threadIdx.y][threadIdx.x] = A[row * n + phase * TILE_WIDTH + threadIdx.x];
        s_B[threadIdx.y][threadIdx.x] = B[(phase * TILE_WIDTH + threadIdx.y) * p + threadIdx.x];
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++) value += s_A[threadIdx.y][i] * s_B[i][threadIdx.x];
        __syncthreads();
    }

    C[idx] = value;
}
