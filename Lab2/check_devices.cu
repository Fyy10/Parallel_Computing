#include "hip/hip_runtime.h"
#include "stdio.h"

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Num of devices: %d\n", deviceCount);
    for (int device = 0; device < deviceCount; device++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has capability %d.%d\n", device, deviceProp.major, deviceProp.minor);
    }
    return 0;
}
